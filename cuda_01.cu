/* A simple CUDA code to learn the memory work between cpu and gpu. */


#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define DATA_SIZE 1024000

int data[DATA_SIZE];
void generate_numbers(int *number, int size)
{
    for(int i = 0; i < size; i++) {
        number[i] = rand() % 50;                                // get random numbers in range of 0 ~ 49
    }
}


__global__ static void sum_of_squares(int *num, int* result)    // declare with __global__ will let function run in gpu, but it only can use void type
{
    int gpu_sum = 0;
    int i;
    for(i = 0; i < DATA_SIZE; i++) {
        gpu_sum += num[i] * num[i];
    }

    *result = gpu_sum;
}


int main()                                              
{
    generate_numbers(data, DATA_SIZE);

    int *gpu_data, *result, gpu_sum, cpu_sum = 0;
    hipMalloc((void**) &gpu_data, sizeof(int) * DATA_SIZE);
    hipMalloc((void**) &result, sizeof(int));
    hipMemcpy(gpu_data, data, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice); // cpu to gpu

    sum_of_squares<<<1, 1, 0>>>(gpu_data, result);                               // <<<numbers of block, numbers of thread, sizeof shared memory>>>(var...); 

    hipMemcpy(&gpu_sum, result, sizeof(int), hipMemcpyDeviceToHost);           // gpu to cpu
   

    // cpu check data
    for(int i = 0; i < DATA_SIZE; i++) {
        cpu_sum += data[i] * data[i];
    }


    printf("gpu_sum: %d\n", gpu_sum);
    printf("cpu_sum: %d\n", cpu_sum);
    
    hipFree(gpu_data);
    hipFree(result);
    return 0;
}
